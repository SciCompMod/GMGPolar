#include "../../../include/ExtrapolatedSmoother/ExtrapolatedSmootherTakeGPU/extrapolatedSmoother.h"

void ExtrapolatedSmootherTakeGPU::extrapolatedSmoothingInPlace(GPU_Vector<double>& x, const GPU_Vector<double>& rhs, GPU_Vector<double>& temp)
{
    const PolarGrid& grid = level_.grid();

    assert(x.size() == grid.numberOfNodes());
    assert(rhs.size() == grid.numberOfNodes());
    assert(temp.size() == grid.numberOfNodes());

    DomainGeometry* device_domain_geometry;
    hipMalloc(&device_domain_geometry, sizeof(DomainGeometry));
    hipMemcpy(device_domain_geometry, &domain_geometry_, sizeof(DomainGeometry), hipMemcpyHostToDevice);

    /* We use precomputed DensityProfileCoefficients values. */
    // DensityProfileCoefficients* device_density_profile;
    // hipMalloc(&device_density_profile, sizeof(DensityProfileCoefficients));
    // hipMemcpy(device_density_profile, &density_profile_coefficients_, sizeof(DensityProfileCoefficients), hipMemcpyHostToDevice);
    
    applyAscOrtho_BlackCircle(x, rhs, temp, device_domain_geometry);
    solveAsc_BlackCircle(x, rhs, temp); 

    applyAscOrtho_WhiteCircle(x, rhs, temp, device_domain_geometry);
    solveAsc_WhiteCircle(x, rhs, temp);

    applyAscOrtho_BlackRadial(x, rhs, device_domain_geometry);
    solveAsc_BlackRadial(x, rhs);

    applyAscOrtho_WhiteRadial(x, rhs, device_domain_geometry);
    solveAsc_WhiteRadial(x, rhs);

    /* We use precomputed DensityProfileCoefficients values. */
    hipFree(device_domain_geometry);
    // hipFree(device_density_profile);
}