#include "hip/hip_runtime.h"
#include "../../../include/Smoother/SmootherTakeGPU/smoother.h"

__global__ void applyAscOrtho_Circle_kernel(
    double* x, double* rhs, double* temp,
    double* circle_lower_diagonals, double* circle_main_diagonals, double* circle_upper_diagonals,
    double* sherman_morrison_gammas,
    PolarGrid* grid, bool DirBC_Interior,
    int start_i_r,
    DomainGeometry* domain_geometry,
    double* coeff_alpha_cache, double* coeff_beta_cache,
    double* sin_theta_cache, double* cos_theta_cache) 
{
    /* The thread block covers a 14x14 region within a 16x16 shared memory block (1-cell halo). */
    const int global_i_r = blockIdx.x * 14 + threadIdx.x - 1;
    const int global_i_theta = blockIdx.y * 14 + threadIdx.y - 1;

    /* Adjust for across origin and periodic boundary. */
    int i_r = global_i_r;
    int i_theta = global_i_theta;
    if(i_r == -1 && !DirBC_Interior){
        i_r = 0;
        i_theta += grid->ntheta() / 2;
    }
    i_theta = grid->wrapThetaIndex(i_theta);

    /* Define bounds for valid global indices (domain + halo). */
    const int min_i_r = DirBC_Interior ? 0 : -1; 
    const int max_i_r = grid->numberSmootherCircles();
    const int min_i_theta = -1; 
    const int max_i_theta = grid->ntheta();

    /* Exit if outside of the computational domain and halo region. */
    if (global_i_r < min_i_r || global_i_r > max_i_r || global_i_theta < min_i_theta || global_i_theta > max_i_theta) return;

    /* Allocate shared memory with padding for avoiding bank conflicts. */
    __shared__ double s_x[16][16 + 1];
    __shared__ double s_arr[16][16 + 1];
    __shared__ double s_art[16][16 + 1];

    /* Local (shared memory) thread indices. */
    const int s_i_r = threadIdx.x;
    const int s_i_theta = threadIdx.y;

    /* Load x value into shared memory. */
    const int center_index = grid->index(i_r, i_theta);
    s_x[s_i_r][s_i_theta] = x[center_index];
    
    /* Compute Jacobian on current node */
    const double r = grid->radius(i_r);
    const double theta = grid->theta(i_theta);

    const double sin_theta = sin_theta_cache[i_theta];
    const double cos_theta = cos_theta_cache[i_theta];
    
    const double Jrr = domain_geometry->dFx_dr(r, theta, sin_theta, cos_theta);
    const double Jtr = domain_geometry->dFy_dr(r, theta, sin_theta, cos_theta);
    const double Jrt = domain_geometry->dFx_dt(r, theta, sin_theta, cos_theta);
    const double Jtt = domain_geometry->dFy_dt(r, theta, sin_theta, cos_theta);

    const double coeff_alpha = coeff_alpha_cache[i_r];

    const double detDF = Jrr * Jtt - Jrt * Jtr;
    const double arr = 0.5 * (Jtt * Jtt + Jrt * Jrt) * coeff_alpha / fabs(detDF);
    const double art = (- Jtt * Jtr - Jrt * Jrr) * coeff_alpha / fabs(detDF);

    /* Share data to nodes in local grid block. */
    s_arr[s_i_r][s_i_theta] = arr;
    s_art[s_i_r][s_i_theta] = art;

    __syncthreads();

    /* Node lies outside of the circle section. */
    if(global_i_r < 0 || global_i_r >= grid->numberSmootherCircles() || global_i_theta < 0 || global_i_theta >= grid->ntheta()) return;
    /* Node lies on the halo. */
    if (s_i_r == 0 || s_i_r == 15 || s_i_theta == 0 || s_i_theta == 15) return;

    /* Node color and smoother color doesnt match. */
    if(i_r % 2 != start_i_r) return;


    /* Prepare temp for the 2nd solution in the Shermann-Morrison formula. */
    if(i_r > 0){
        int matrix_index = i_r * grid->ntheta();
        if(i_theta == 0){
            temp[matrix_index + i_theta] = sherman_morrison_gammas[i_r];
        }
        else if(i_theta > 0 && i_theta < grid->ntheta()-1){
            temp[matrix_index + i_theta] = 0.0;
        }
        else if(i_theta == grid->ntheta()-1){
            temp[matrix_index + i_theta] = circle_upper_diagonals[matrix_index + grid->ntheta() - 1];
        }
    }

    /* Compute neighbor distances */
    bool isOnInnerBoundary = (i_r == 0);

    double h1 = DirBC_Interior ? 
        ((!isOnInnerBoundary) ? grid->radialSpacing(i_r - 1) : 0.0) :
        ((!isOnInnerBoundary) ? grid->radialSpacing(i_r - 1) : 2.0 * grid->radius(0));
    double h2 = grid->radialSpacing(i_r);
    double k1 = grid->angularSpacing(i_theta - 1);                                                          
    double k2 = grid->angularSpacing(i_theta);

    if (!isOnInnerBoundary) {   

        double coeff1 = 0.5 * (k1 + k2) / h1;
        double coeff2 = 0.5 * (k1 + k2) / h2;

        x[center_index] = rhs[center_index] - (
            - coeff1 * (s_arr[s_i_r][s_i_theta] + s_arr[s_i_r-1][s_i_theta]) * s_x[s_i_r-1][s_i_theta] /* Left */  
            - coeff2 * (s_arr[s_i_r][s_i_theta] + s_arr[s_i_r+1][s_i_theta]) * s_x[s_i_r+1][s_i_theta] /* Right */   

            - 0.25 * (s_art[s_i_r-1][s_i_theta] + s_art[s_i_r][s_i_theta-1]) * s_x[s_i_r-1][s_i_theta-1] /* Bottom Left */
            + 0.25 * (s_art[s_i_r+1][s_i_theta] + s_art[s_i_r][s_i_theta-1]) * s_x[s_i_r+1][s_i_theta-1] /* Bottom Right */
            + 0.25 * (s_art[s_i_r-1][s_i_theta] + s_art[s_i_r][s_i_theta+1]) * s_x[s_i_r-1][s_i_theta+1] /* Top Left */
            - 0.25 * (s_art[s_i_r+1][s_i_theta] + s_art[s_i_r][s_i_theta+1]) * s_x[s_i_r+1][s_i_theta+1] /* Top Right */
        );
    }
    else if(isOnInnerBoundary && !DirBC_Interior){

        double coeff2 = 0.5 * (k1 + k2) / h2;

        x[center_index] = rhs[center_index] - (
            - coeff2 * (s_arr[s_i_r][s_i_theta] + s_arr[s_i_r+1][s_i_theta]) * s_x[s_i_r+1][s_i_theta] /* Right */

            + 0.25 * (s_art[s_i_r+1][s_i_theta] + s_art[s_i_r][s_i_theta-1]) *  s_x[s_i_r+1][s_i_theta-1] /* Bottom Right */
            - 0.25 * (s_art[s_i_r+1][s_i_theta] + s_art[s_i_r][s_i_theta+1]) * s_x[s_i_r+1][s_i_theta+1] /* Top Right */  
        );
    }
    else if(isOnInnerBoundary && DirBC_Interior){
        x[center_index] = rhs[center_index];           
    }
}



void SmootherTakeGPU::applyAscOrtho_BlackCircle(
    GPU_Vector<double>& x, const GPU_Vector<double>& rhs, GPU_Vector<double>& temp, 
    DomainGeometry* device_domain_geometry)
{

    const PolarGrid& grid = level_.grid();

    const GPU_Vector<double>& sin_theta_cache = level_.levelCache().GPU_sin_theta();
    const GPU_Vector<double>& cos_theta_cache = level_.levelCache().GPU_cos_theta();

    const GPU_Vector<double>& coeff_alpha_cache = level_.levelCache().GPU_coeff_alpha();
    const GPU_Vector<double>& coeff_beta_cache = level_.levelCache().GPU_coeff_beta();

    const int start_black_circles = (grid.numberSmootherCircles() % 2 == 0) ? 1 : 0;

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((grid.numberSmootherCircles() + 14 - 1) / 14, (grid.ntheta() + 14 - 1) / 14);
    applyAscOrtho_Circle_kernel<<<numBlocks, threadsPerBlock>>>(
        x.data(), rhs.data(), temp.data(),
        circle_lower_diagonals_, circle_main_diagonals_, circle_upper_diagonals_,
        sherman_morrison_gammas_,
        level_.device_grid(), DirBC_Interior_,
        start_black_circles,
        device_domain_geometry, 
        coeff_alpha_cache.data(), coeff_beta_cache.data(), 
        sin_theta_cache.data(), cos_theta_cache.data()
    );
    hipDeviceSynchronize();
}



void SmootherTakeGPU::applyAscOrtho_WhiteCircle(
    GPU_Vector<double>& x, const GPU_Vector<double>& rhs, GPU_Vector<double>& temp, 
    DomainGeometry* device_domain_geometry)
{
    const PolarGrid& grid = level_.grid();

    const GPU_Vector<double>& sin_theta_cache = level_.levelCache().GPU_sin_theta();
    const GPU_Vector<double>& cos_theta_cache = level_.levelCache().GPU_cos_theta();

    const GPU_Vector<double>& coeff_alpha_cache = level_.levelCache().GPU_coeff_alpha();
    const GPU_Vector<double>& coeff_beta_cache = level_.levelCache().GPU_coeff_beta();

    const int start_white_circles = (grid.numberSmootherCircles() % 2 == 0) ? 0 : 1;

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((grid.numberSmootherCircles() + 14 - 1) / 14, (grid.ntheta() + 14 - 1) / 14);
    applyAscOrtho_Circle_kernel<<<numBlocks, threadsPerBlock>>>(
        x.data(), rhs.data(), temp.data(),
        circle_lower_diagonals_, circle_main_diagonals_, circle_upper_diagonals_,
        sherman_morrison_gammas_,
        level_.device_grid(), DirBC_Interior_,
        start_white_circles,
        device_domain_geometry, 
        coeff_alpha_cache.data(), coeff_beta_cache.data(), 
        sin_theta_cache.data(), cos_theta_cache.data()
    );
    hipDeviceSynchronize();
}
