#include "hip/hip_runtime.h"
#include "../../../include/Smoother/SmootherTakeGPU/smoother.h"

/* Constructor */

SmootherTakeGPU::SmootherTakeGPU(const Level& level, const DomainGeometry& domain_geometry,
                   const DensityProfileCoefficients& density_profile_coefficients, bool DirBC_Interior)
    /* Constructor Members */
    : level_(level)
    , domain_geometry_(domain_geometry)
    , density_profile_coefficients_(density_profile_coefficients)
    , DirBC_Interior_(DirBC_Interior)
    /* Circle Tridiagonal Matrices */
    , circle_main_diagonals_(nullptr)
    , circle_lower_diagonals_(nullptr)
    , circle_upper_diagonals_(nullptr)
    , sherman_morrison_gammas_(nullptr)
    /* Radial Tridiagonal Matrices */
    , radial_main_diagonals_(nullptr)
    , radial_lower_diagonals_(nullptr)
    , radial_upper_diagonals_(nullptr)
    /* Tridiagonal Solver Buffer */
    , pBuffer_(nullptr)
    /* Inner Boundary CSR Matrix */
    , csrValA_(nullptr)
    , csrRowPtrA_(nullptr)
    , csrColIndA_(nullptr)
    /* Inner Boundary Mumps COO Matrix */
    , inner_boundary_matrix_row_indices_(nullptr)
    , inner_boundary_matrix_column_indices_(nullptr)
    , inner_boundary_matrix_values_(nullptr)
    , d_inner_boundary_matrix_row_indices_(nullptr)
    , d_inner_boundary_matrix_column_indices_(nullptr)
    , d_inner_boundary_matrix_values_(nullptr)
{
    const PolarGrid& grid = level.grid();

    int nr = grid.nr();
    int ntheta = grid.ntheta();
    int number_smoother_circles = grid.numberSmootherCircles();
    int length_smoother_radial = grid.lengthSmootherRadial();

    int circle_batch_count = number_smoother_circles;
    int circle_m = ntheta;
    /* Cyclic Tridiagonal Circle Matrices */
    hipMalloc(&circle_lower_diagonals_, circle_m * circle_batch_count * sizeof(double));
    hipMemset(circle_lower_diagonals_, 0, circle_m * circle_batch_count * sizeof(double));
    hipMalloc(&circle_main_diagonals_, circle_m * circle_batch_count * sizeof(double));
    hipMemset(circle_main_diagonals_, 0, circle_m * circle_batch_count * sizeof(double));
    hipMalloc(&circle_upper_diagonals_, circle_m * circle_batch_count * sizeof(double));
    hipMemset(circle_upper_diagonals_, 0, circle_m * circle_batch_count * sizeof(double));
    /* Cuda does not supply a cyclic tridiagonal solver. */
    /* Thus we use the Sherman–Morrison formula to reduce the problem to a simple tridiagonal problem with two right hand sides. */
    hipMalloc(&sherman_morrison_gammas_, circle_batch_count * sizeof(double));
    hipMemset(sherman_morrison_gammas_, 0, circle_batch_count * sizeof(double));
    hipMalloc(&factor_, circle_batch_count * sizeof(double));
    hipMemset(factor_, 0, circle_batch_count * sizeof(double));
    /* Remark: The 1st cylic tridiagonal matrix on the interior boundary is unused. */

    int radial_batch_count = ntheta;
    int radial_m = length_smoother_radial;
    /* Tridiagonal Radial Matrices */
    hipMalloc(&radial_lower_diagonals_, radial_m * radial_batch_count * sizeof(double));
    hipMemset(radial_lower_diagonals_, 0, radial_m * radial_batch_count * sizeof(double));
    hipMalloc(&radial_main_diagonals_, radial_m * radial_batch_count * sizeof(double));
    hipMemset(radial_main_diagonals_, 0, radial_m * radial_batch_count * sizeof(double));
    hipMalloc(&radial_upper_diagonals_, radial_m * radial_batch_count * sizeof(double));
    hipMemset(radial_upper_diagonals_, 0, radial_m * radial_batch_count * sizeof(double));

    /* Tridiaginal Cuda Solver */
    hipsparseCreate(&sparse_handle_);
    /* General Matrix Cuda Solver */
    hipsolverSpCreate(&solver_handle_);
    hipsparseCreateMatDescr(&descrA_);
    hipsparseSetMatType(descrA_, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA_, HIPSPARSE_INDEX_BASE_ZERO);

    int interior_boundary_matrix_m = ntheta;
    int interior_boundary_matrix_nnz = DirBC_Interior_ ? ntheta : 4 * ntheta;
    /* Interior Boundary CSR Matrix */
    hipMalloc(&csrValA_, interior_boundary_matrix_nnz * sizeof(double));
    hipMemset(csrValA_, 0, interior_boundary_matrix_nnz * sizeof(double));
    hipMalloc(&csrRowPtrA_, (interior_boundary_matrix_m + 1) * sizeof(int));
    hipMemset(csrRowPtrA_, 0, (interior_boundary_matrix_m + 1) * sizeof(int));
    hipMalloc(&csrColIndA_, interior_boundary_matrix_nnz * sizeof(int));
    hipMemset(csrColIndA_, 0, interior_boundary_matrix_nnz * sizeof(int));

    /* Allocate Tridiagonal Solver Buffer. */
    size_t bufferSizeInBytes_Circle;
    hipsparseDgtsv2StridedBatch_bufferSizeExt(
        sparse_handle_, ntheta, nullptr, nullptr, nullptr, nullptr, 
        (number_smoother_circles+1) / 2, 2 * ntheta, &bufferSizeInBytes_Circle);

    size_t bufferSizeInBytes_Radial;
    hipsparseDgtsv2StridedBatch_bufferSizeExt(
        sparse_handle_, length_smoother_radial, nullptr, nullptr, nullptr, nullptr, 
        ntheta / 2, 2 * length_smoother_radial, &bufferSizeInBytes_Radial);

    /* The Tridiagonal solvers require 5 * batch_count * m * sizeof(double) bytes. */
    /* Alternatively use four different pBuffers for each tridiagonal batch solver. */
    size_t max_pBufferSizeInBytes = std::max(bufferSizeInBytes_Circle, bufferSizeInBytes_Radial);
    hipMalloc(&pBuffer_, max_pBufferSizeInBytes);

    /* Inner Boundary Mumps COO Matrix */
    int nnz = DirBC_Interior_ ? grid.ntheta() : 4 * grid.ntheta(); 
    inner_boundary_matrix_row_indices_ = std::make_unique<int[]>(nnz);
    inner_boundary_matrix_column_indices_ = std::make_unique<int[]>(nnz);
    inner_boundary_matrix_values_ = std::make_unique<double[]>(nnz);

    hipMalloc(&d_inner_boundary_matrix_row_indices_, nnz * sizeof(int));
    hipMemset(d_inner_boundary_matrix_row_indices_, 0, nnz* sizeof(int));
    hipMalloc(&d_inner_boundary_matrix_column_indices_, nnz * sizeof(int));
    hipMemset(d_inner_boundary_matrix_column_indices_, 0, nnz* sizeof(int));
    hipMalloc(&d_inner_boundary_matrix_values_, nnz * sizeof(double));
    hipMemset(d_inner_boundary_matrix_values_, 0, nnz* sizeof(double));

    /* Build Smoother Matrices we have allocated. */
    buildAscMatrices();

    initializeMumps();

    /* The cyclic tridiagonal Matrices need to be adjusted to a system of a non-cyclic tridiagonal matrices. */
    adjustAscCircle_ShermanMorrison();
}



/* Destructor */

SmootherTakeGPU::~SmootherTakeGPU() {
    /* Cyclic Tridiagonal Circle Matrices */
    if (circle_lower_diagonals_) {
        hipFree(circle_lower_diagonals_);
        circle_lower_diagonals_ = nullptr;
    }
    if (circle_main_diagonals_) {
        hipFree(circle_main_diagonals_);
        circle_main_diagonals_ = nullptr;
    }
    if (circle_upper_diagonals_) {
        hipFree(circle_upper_diagonals_);
        circle_upper_diagonals_ = nullptr;
    }
    /* Cuda does not supply a cyclic tridiagonal solver. */
    /* Thus we use the Sherman–Morrison formula to reduce the problem to a simple tridiagonal problem with two right hand sides. */
    if (sherman_morrison_gammas_) {
        hipFree(sherman_morrison_gammas_);
        sherman_morrison_gammas_ = nullptr;
    }
    if (factor_) {
        hipFree(factor_);
        factor_ = nullptr;
    }

    /* Tridiagonal Radial Matrices */
    if (radial_lower_diagonals_) {
        hipFree(radial_lower_diagonals_);
        radial_lower_diagonals_ = nullptr;
    }
    if (radial_main_diagonals_) {
        hipFree(radial_main_diagonals_);
        radial_main_diagonals_ = nullptr;
    }
    if (radial_upper_diagonals_) {
        hipFree(radial_upper_diagonals_);
        radial_upper_diagonals_ = nullptr;
    }

    /* Tridiaginal Cuda Solver */
    hipsparseDestroy(sparse_handle_);
    /* General Matrix Cuda Solver */
    hipsolverSpDestroy(solver_handle_);
    hipsparseDestroyMatDescr(descrA_);

    /* Interior Boundary CSR Matrix */
    if (csrValA_) {
        hipFree(csrValA_);
        csrValA_ = nullptr;
    }
    if (csrRowPtrA_) {
        hipFree(csrRowPtrA_);
        csrRowPtrA_ = nullptr;
    }
    if (csrColIndA_) {
        hipFree(csrColIndA_);
        csrColIndA_ = nullptr;
    }

    /* Free Tridiagonal Solver Buffer. */
    if (pBuffer_) {
        hipFree(pBuffer_);
        pBuffer_ = nullptr;
    }

    /* Inner Boundary Mumps COO Matrix */
    if (d_inner_boundary_matrix_row_indices_) {
        hipFree(d_inner_boundary_matrix_row_indices_);
        d_inner_boundary_matrix_row_indices_ = nullptr;
    }
    if (d_inner_boundary_matrix_column_indices_) {
        hipFree(d_inner_boundary_matrix_column_indices_);
        d_inner_boundary_matrix_column_indices_ = nullptr;
    }
    if (d_inner_boundary_matrix_values_) {
        hipFree(d_inner_boundary_matrix_values_);
        d_inner_boundary_matrix_values_ = nullptr;
    }

    finalizeMumpsSolver();
}