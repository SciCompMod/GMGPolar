#include "../../include/Level/level.h"

#include "../../include/Residual/ResidualTakeCPU/residual.h"
#include "../../include/Residual/ResidualTakeGPU/residual.h"

#include "../../include/DirectSolver/directSolver.h"

#include "../../include/Smoother/SmootherTakeCPU/smoother.h"
#include "../../include/Smoother/SmootherTakeGPU/smoother.h"

#include "../../include/ExtrapolatedSmoother/ExtrapolatedSmootherTakeCPU/extrapolatedSmoother.h"
#include "../../include/ExtrapolatedSmoother/ExtrapolatedSmootherTakeGPU/extrapolatedSmoother.h"

Level::Level(const int level, const ProcessingType processing_type, std::unique_ptr<const PolarGrid> grid,
             std::unique_ptr<const LevelCache> level_cache, const ExtrapolationType extrapolation, const bool FMG)
    : level_(level)
    , processing_type_(processing_type)
    , grid_(std::move(grid))
    , level_cache_(std::move(level_cache))
    , extrapolation_(extrapolation)
    , FMG_(FMG)
    , device_grid_(nullptr)
{
    const auto num_nodes = grid_->numberOfNodes();
    const auto rhs_size =
        (FMG_ || level_ == 0 || (level_ == 1 && extrapolation_ != ExtrapolationType::NONE)) ? num_nodes : 0;
    const auto error_size = (level_ > 0) ? num_nodes : 0;

    if (processing_type == ProcessingType::CPU || processing_type == ProcessingType::CPU_HYBRID) {
        rhs_              = Vector<double>(rhs_size);
        solution_         = Vector<double>(num_nodes);
        residual_         = Vector<double>(num_nodes);
        error_correction_ = Vector<double>(error_size);
    }
    else {
        rhs_              = Vector<double>(0);
        solution_         = Vector<double>(0);
        residual_         = Vector<double>(0);
        error_correction_ = Vector<double>(0);
    }

    if (processing_type == ProcessingType::GPU || processing_type == ProcessingType::CPU_HYBRID) {
        hipMalloc(&device_grid_, sizeof(PolarGrid));
        hipMemcpy(device_grid_, grid_.get(), sizeof(PolarGrid), hipMemcpyHostToDevice);
        gpu_rhs_              = GPU_Vector<double>(rhs_size);
        gpu_solution_         = GPU_Vector<double>(num_nodes);
        gpu_residual_         = GPU_Vector<double>(num_nodes);
        gpu_error_correction_ = GPU_Vector<double>(error_size);
    }
    else {
        gpu_rhs_              = GPU_Vector<double>(0);
        gpu_solution_         = GPU_Vector<double>(0);
        gpu_residual_         = GPU_Vector<double>(0);
        gpu_error_correction_ = GPU_Vector<double>(0);
    }
}

Level::Level(Level&& other) noexcept
    : level_(other.level_)
    , processing_type_(other.processing_type_)
    , grid_(std::move(other.grid_))
    , device_grid_(other.device_grid_)
    , level_cache_(std::move(other.level_cache_))
    , extrapolation_(other.extrapolation_)
    , FMG_(other.FMG_)
    , rhs_(std::move(other.rhs_))
    , solution_(std::move(other.solution_))
    , residual_(std::move(other.residual_))
    , error_correction_(std::move(other.error_correction_))
    , gpu_rhs_(std::move(other.gpu_rhs_))
    , gpu_solution_(std::move(other.gpu_solution_))
    , gpu_residual_(std::move(other.gpu_residual_))
    , gpu_error_correction_(std::move(other.gpu_error_correction_))
{
    other.device_grid_ = nullptr;
}

// Move Assignment Operator
Level& Level::operator=(Level&& other) noexcept
{
    if (this != &other) {
        // Free existing GPU resources if necessary
        if (device_grid_) {
            hipFree(device_grid_);
        }

        // Move data members
        level_         = other.level_;
        grid_          = std::move(other.grid_);
        device_grid_   = other.device_grid_;
        level_cache_   = std::move(other.level_cache_);
        extrapolation_ = other.extrapolation_;
        FMG_           = other.FMG_;

        rhs_                  = std::move(other.rhs_);
        solution_             = std::move(other.solution_);
        residual_             = std::move(other.residual_);
        error_correction_     = std::move(other.error_correction_);
        gpu_rhs_              = std::move(other.gpu_rhs_);
        gpu_solution_         = std::move(other.gpu_solution_);
        gpu_residual_         = std::move(other.gpu_residual_);
        gpu_error_correction_ = std::move(other.gpu_error_correction_);

        // Leave the source object in a valid state
        other.device_grid_ = nullptr;
    }
    return *this;
}
// Destructor
Level::~Level()
{
if (device_grid_) {
    hipPointerAttribute_t attributes;
    hipError_t err = hipPointerGetAttributes(&attributes, device_grid_);
    if (err != hipSuccess) {
        std::cerr << "Invalid device pointer before free: " << hipGetErrorString(err) << std::endl;
    }
    hipFree(device_grid_);
    device_grid_ = nullptr;
}
}

// ---------------- //
// Getter Functions //
int Level::level() const
{
    return level_;
}

const PolarGrid& Level::grid() const
{
    return *grid_;
}

ProcessingType Level::processingType() const
{
    return processing_type_;
}

PolarGrid* Level::device_grid() const
{
    return device_grid_;
}

const LevelCache& Level::levelCache() const
{
    return *level_cache_;
}

Vector<double>& Level::rhs()
{
    return rhs_;
}
const Vector<double>& Level::rhs() const
{
    return rhs_;
}
Vector<double>& Level::solution()
{
    return solution_;
}
const Vector<double>& Level::solution() const
{
    return solution_;
}
Vector<double>& Level::residual()
{
    return residual_;
}
const Vector<double>& Level::residual() const
{
    return residual_;
}
Vector<double>& Level::error_correction()
{
    return error_correction_;
}
const Vector<double>& Level::error_correction() const
{
    return error_correction_;
}

GPU_Vector<double>& Level::GPU_rhs()
{
    return gpu_rhs_;
}
const GPU_Vector<double>& Level::GPU_rhs() const
{
    return gpu_rhs_;
}
GPU_Vector<double>& Level::GPU_solution()
{
    return gpu_solution_;
}
const GPU_Vector<double>& Level::GPU_solution() const
{
    return gpu_solution_;
}
GPU_Vector<double>& Level::GPU_residual()
{
    return gpu_residual_;
}
const GPU_Vector<double>& Level::GPU_residual() const
{
    return gpu_residual_;
}
GPU_Vector<double>& Level::GPU_error_correction()
{
    return gpu_error_correction_;
}
const GPU_Vector<double>& Level::GPU_error_correction() const
{
    return gpu_error_correction_;
}


// -------------- //
// Apply Residual //
void Level::initializeResidual(const DomainGeometry& domain_geometry,
                               const DensityProfileCoefficients& density_profile_coefficients,
                               const bool DirBC_Interior)
{
    if(processing_type_ == ProcessingType::GPU){
        op_residual_GPU_ = std::make_unique<ResidualTakeGPU>(*this, domain_geometry, density_profile_coefficients, DirBC_Interior);
        if (!op_residual_GPU_) throw std::runtime_error("Failed to initialize GPU Residual.");
    }
    else{
        op_residual_CPU_ = std::make_unique<ResidualTakeCPU>(*this, domain_geometry, density_profile_coefficients, DirBC_Interior);
        if (!op_residual_CPU_) throw std::runtime_error("Failed to initialize CPU Residual.");
    }
}

void Level::computeResidual(Vector<double>& result, const Vector<double>& rhs, const Vector<double>& x) const
{
    if (!op_residual_CPU_) throw std::runtime_error("CPU Residual not initialized.");
    op_residual_CPU_->computeResidual(result, rhs, x);
}
void Level::computeResidual(GPU_Vector<double>& result, const GPU_Vector<double>& rhs, const GPU_Vector<double>& x) const
{
    if (!op_residual_GPU_) throw std::runtime_error("GPU Residual not initialized.");
    op_residual_GPU_->computeResidual(result, rhs, x);
}


// ------------------- //
// Solve coarse System //
void Level::initializeDirectSolver(const DomainGeometry& domain_geometry,
                                   const DensityProfileCoefficients& density_profile_coefficients,
                                   const bool DirBC_Interior)
{

    op_directSolver_ = std::make_unique<DirectSolver>(*this, domain_geometry, density_profile_coefficients, DirBC_Interior);
    if (!op_directSolver_) throw std::runtime_error("Failed to initialize Direct Solver.");
}
void Level::directSolveInPlace(Vector<double>& x) const
{
    if (!op_directSolver_) throw std::runtime_error("Coarse Solver not initialized.");
    op_directSolver_->solveInPlace(x);
}


// --------------- //
// Apply Smoothing //
void Level::initializeSmoothing(const DomainGeometry& domain_geometry,
                                const DensityProfileCoefficients& density_profile_coefficients,
                                const bool DirBC_Interior)
{
    if(processing_type_ == ProcessingType::GPU){
        op_smoother_GPU_ = std::make_unique<SmootherTakeGPU>(*this, domain_geometry, density_profile_coefficients, DirBC_Interior);
        if (!op_smoother_GPU_) throw std::runtime_error("Failed to initialize GPU Smoother.");
    }
    else{
        op_smoother_CPU_ = std::make_unique<SmootherTakeCPU>(*this, domain_geometry, density_profile_coefficients, DirBC_Interior);
        if (!op_smoother_CPU_) throw std::runtime_error("Failed to initialize CPU Smoother.");
    }
}

void Level::smoothingInPlace(Vector<double>& x, const Vector<double>& rhs, Vector<double>& temp) const
{
    if (!op_smoother_CPU_) throw std::runtime_error("CPU Smoother not initialized.");
    op_smoother_CPU_->smoothingInPlace(x, rhs, x);
}
void Level::smoothingInPlace(GPU_Vector<double>& x, const GPU_Vector<double>& rhs, GPU_Vector<double>& temp) const
{
    if (!op_smoother_GPU_) throw std::runtime_error("GPU Smoother not initialized.");
    op_smoother_GPU_->smoothingInPlace(x, rhs, temp);
}

// ---------------------------- //
// Apply Extrapolated Smoothing //
void Level::initializeExtrapolatedSmoothing(const DomainGeometry& domain_geometry,
                                const DensityProfileCoefficients& density_profile_coefficients,
                                const bool DirBC_Interior)
{
    if(processing_type_ == ProcessingType::GPU){
        op_extrapolated_smoother_GPU_ = std::make_unique<ExtrapolatedSmootherTakeGPU>(*this, domain_geometry, density_profile_coefficients, DirBC_Interior);
        if (!op_extrapolated_smoother_GPU_) throw std::runtime_error("Failed to initialize GPU Extrapolated Smoother.");
    }
    else{
        op_extrapolated_smoother_CPU_ = std::make_unique<ExtrapolatedSmootherTakeCPU>(*this, domain_geometry, density_profile_coefficients, DirBC_Interior);
        if (!op_extrapolated_smoother_CPU_) throw std::runtime_error("Failed to initialize CPU Extrapolated Smoother.");
    }
}

void Level::extrapolatedSmoothingInPlace(Vector<double>& x, const Vector<double>& rhs, Vector<double>& temp) const
{
    if (!op_extrapolated_smoother_CPU_) throw std::runtime_error("CPU Extrapolated Smoother not initialized.");
    op_extrapolated_smoother_CPU_->extrapolatedSmoothingInPlace(x, rhs, x);
}
void Level::extrapolatedSmoothingInPlace(GPU_Vector<double>& x, const GPU_Vector<double>& rhs, GPU_Vector<double>& temp) const
{
    if (!op_extrapolated_smoother_GPU_) throw std::runtime_error("GPU Extrapolated Smoother not initialized.");
    op_extrapolated_smoother_GPU_->extrapolatedSmoothingInPlace(x, rhs, temp);
}

