#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <vector>
#include <cmath>

#define CHECK_CUDA(call)                                                                 \
    {                                                                                   \
        hipError_t err = call;                                                         \
        if (err != hipSuccess) {                                                       \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": "        \
                      << hipGetErrorString(err) << std::endl;                          \
            exit(EXIT_FAILURE);                                                         \
        }                                                                               \
    }

#define CHECK_CUSPARSE(call)                                                            \
    {                                                                                   \
        hipsparseStatus_t err = call;                                                    \
        if (err != HIPSPARSE_STATUS_SUCCESS) {                                           \
            std::cerr << "cuSPARSE error in " << __FILE__ << ":" << __LINE__ << ": "    \
                      << err << std::endl;                                              \
            exit(EXIT_FAILURE);                                                         \
        }                                                                               \
    }






void test_cusparseDgtsv2StridedBatch_2(int m, int batchCount) {
    int batchStride = m; // Minimum batch stride

    // Allocate memory for diagonals and RHS
    size_t diag_size = batchCount * batchStride * sizeof(double);
    double *dl, *d, *du, *x, *x_2;
    CHECK_CUDA(hipMalloc(&dl, diag_size));
    CHECK_CUDA(hipMalloc(&d, diag_size));
    CHECK_CUDA(hipMalloc(&du, diag_size));
    CHECK_CUDA(hipMalloc(&x, diag_size));
    CHECK_CUDA(hipMalloc(&x_2, diag_size));

    // Initialize diagonals and RHS with dummy data
    std::vector<double> h_dl(batchCount * batchStride, -1.0);
    std::vector<double> h_d(batchCount * batchStride, 4.0);
    std::vector<double> h_du(batchCount * batchStride, -1.0);
    std::vector<double> h_x(batchCount * batchStride, 1.0);
    std::vector<double> h_x_2(batchCount * batchStride, 2.0);


    CHECK_CUDA(hipMemcpy(dl, h_dl.data(), diag_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d, h_d.data(), diag_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(du, h_du.data(), diag_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(x, h_x.data(), diag_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(x_2, h_x_2.data(), diag_size, hipMemcpyHostToDevice));


    // Create cuSPARSE handle
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Query buffer size
    void *pBuffer = nullptr;
    size_t pBufferSizeInBytes;
    CHECK_CUSPARSE(hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, dl, d, du, nullptr, batchCount, batchStride, &pBufferSizeInBytes));
    CHECK_CUDA(hipMalloc(&pBuffer, pBufferSizeInBytes));

    std::cout<<pBufferSizeInBytes<<std::endl;

    // Timing setup
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Solve systems and measure time
    CHECK_CUDA(hipEventRecord(start));
    CHECK_CUSPARSE(hipsparseDgtsv2StridedBatch(handle, m, dl, d, du, x, batchCount, batchStride, pBuffer));
    CHECK_CUSPARSE(hipsparseDgtsv2StridedBatch(handle, m, dl, d, du, x_2, batchCount, batchStride, pBuffer));
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Execution time: " << milliseconds << " ms" << std::endl;

    // Calculate and print norm of x
    std::vector<double> h_x_result(batchCount * batchStride);
    CHECK_CUDA(hipMemcpy(h_x_result.data(), x, diag_size, hipMemcpyDeviceToHost));
    double norm = 0.0;
    for (const double &val : h_x_result) {
        norm += val * val;
    }
    norm = std::sqrt(norm);
    std::cout << "Norm of x: " << norm << std::endl;

    std::vector<double> h_x_result_2(batchCount * batchStride);
    CHECK_CUDA(hipMemcpy(h_x_result_2.data(), x_2, diag_size, hipMemcpyDeviceToHost));
    double norm2 = 0.0;
    for (const double &val : h_x_result_2) {
        norm2 += val * val;
    }
    norm2 = std::sqrt(norm2);
    std::cout << "Norm of x: " << norm2 << std::endl;


    // Clean up
    hipFree(dl);
    hipFree(d);
    hipFree(du);
    hipFree(x);
    hipFree(pBuffer);
    hipsparseDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}







void test_cusparseDgtsv2StridedBatch(int m, int batchCount) {
    int batchStride = m; // Minimum batch stride

    // Allocate memory for diagonals and RHS
    size_t diag_size = batchCount * batchStride * sizeof(double);
    double *dl, *d, *du, *x;
    CHECK_CUDA(hipMalloc(&dl, diag_size));
    CHECK_CUDA(hipMalloc(&d, diag_size));
    CHECK_CUDA(hipMalloc(&du, diag_size));
    CHECK_CUDA(hipMalloc(&x, diag_size));

    // Initialize diagonals and RHS with dummy data
    std::vector<double> h_dl(batchCount * batchStride, -1.0);
    std::vector<double> h_d(batchCount * batchStride, 4.0);
    std::vector<double> h_du(batchCount * batchStride, -1.0);
    std::vector<double> h_x(batchCount * batchStride, 1.0);


    CHECK_CUDA(hipMemcpy(dl, h_dl.data(), diag_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d, h_d.data(), diag_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(du, h_du.data(), diag_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(x, h_x.data(), diag_size, hipMemcpyHostToDevice));

    // Create cuSPARSE handle
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Query buffer size
    void *pBuffer = nullptr;
    size_t pBufferSizeInBytes;
    CHECK_CUSPARSE(hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m/2, nullptr, nullptr, nullptr, nullptr, batchCount, 2*batchStride, &pBufferSizeInBytes));
    CHECK_CUDA(hipMalloc(&pBuffer, pBufferSizeInBytes));

    std::cout<<pBufferSizeInBytes<<std::endl;
    std::cout<< 5 * batchCount * m * sizeof(double) <<std::endl;



    // Timing setup
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Solve systems and measure time
    CHECK_CUDA(hipEventRecord(start));
    CHECK_CUSPARSE(hipsparseDgtsv2StridedBatch(handle, m, dl, d, du, x, batchCount, batchStride, pBuffer));
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Execution time: " << milliseconds << " ms" << std::endl;

    // Calculate and print norm of x
    std::vector<double> h_x_result(batchCount * batchStride);
    CHECK_CUDA(hipMemcpy(h_x_result.data(), x, diag_size, hipMemcpyDeviceToHost));
    double norm = 0.0;
    for (const double &val : h_x_result) {
        norm += val * val;
    }
    norm = std::sqrt(norm);
    std::cout << "Norm of x: " << norm << std::endl;

    // Clean up
    hipFree(dl);
    hipFree(d);
    hipFree(du);
    hipFree(x);
    hipFree(pBuffer);
    hipsparseDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    const int m = 4096;         // System size
    const int batchCount = 427; // Number of systems

    test_cusparseDgtsv2StridedBatch(m, batchCount * 2);

    // test_cusparseDgtsv2StridedBatch_2(m, batchCount);

    return 0;
}